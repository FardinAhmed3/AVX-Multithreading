#include <iostream>
#include <random>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

__global__ void monteCarloPi(unsigned long long *d_count, unsigned long iterations, unsigned int seed) {
    unsigned long long localCount = 0;
    hiprandState state;
    hiprand_init(seed + threadIdx.x + blockIdx.x * blockDim.x, 0, 0, &state);
    
    for (unsigned long i = 0; i < iterations; i++) {
        double x = hiprand_uniform(&state);
        double y = hiprand_uniform(&state);
        if (x * x + y * y <= 1.0)
            localCount++;
    }
    atomicAdd(d_count, localCount);
}

int main() {
    unsigned long totalIterations;
    std::cout << "Enter total number of iterations: ";
    std::cin >> totalIterations;
    
    int numThreads = 256;
    int numBlocks = 1024;
    unsigned long iterationsPerThread = totalIterations / (numThreads * numBlocks);
    
    unsigned long long *d_count, h_count = 0;
    hipMalloc(&d_count, sizeof(unsigned long long));
    hipMemcpy(d_count, &h_count, sizeof(unsigned long long), hipMemcpyHostToDevice);
    
    monteCarloPi<<<numBlocks, numThreads>>>(d_count, iterationsPerThread, time(0));
    hipMemcpy(&h_count, d_count, sizeof(unsigned long long), hipMemcpyDeviceToHost);
    
    double piEstimate = 4.0 * static_cast<double>(h_count) / totalIterations;
    std::cout << "Estimated Pi = " << piEstimate << std::endl;
    
    hipFree(d_count);
    return 0;
}